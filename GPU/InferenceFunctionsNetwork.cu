#include "hip/hip_runtime.h"
#include "InferenceFunctionsNetwork.h"

using namespace std;
//**********************************************************************************************************************//
//**********************************************************************************************************************//
//
//Authors: This code is originally developed by Isabel Valera for the "General latent feature models for heterogeneous dataset" paper
//It has been modified by Zahra Sheikhbahaee to take into account the network data. In new version, there are two pseudo likelihood, one is related to the heterogeneous attribute data and the other belongs to the adjacency matrix and we posit that there is an affinity matrix which takes into account the degree of homophily/heterophily between the latent communities (features)......
//
//**********************************************************************************************************************//
//**********************************************************************************************************************//

void
SampleY(double missing, int N, int d, int K, char Cd, int Rd, double fd, double mud, double wd, double s2Y, double s2u,
        double s2theta, gsl_matrix *X, gsl_matrix *Z, gsl_matrix *Yd, gsl_matrix *Bd, gsl_vector *thetad,
        const gsl_rng *seed) {
    double sYd = sqrt(s2Y);
    double stheta = sqrt(s2theta);
    gsl_matrix_view Zn;
    gsl_matrix_view Bd_view;
    gsl_matrix *muy;
    double xnd;
    switch (Cd) {
        case 'g':
            //real-valued observations Eq. (14)
            muy = gsl_matrix_calloc(1, 1);
            for (int n = 0; n < N; n++) {
                xnd = gsl_matrix_get(X, d, n);
                Zn = gsl_matrix_submatrix(Z, 0, n, K, 1);
                Bd_view = gsl_matrix_submatrix(Bd, 0, 0, K, 1);
                //muy=Zn*Bd_view
                matrix_multiply(&Zn.matrix, &Bd_view.matrix, muy, 1, 0, CblasTrans, CblasNoTrans);
                if (xnd == missing || gsl_isnan(xnd)) {
                    gsl_matrix_set(Yd, 0, n, gsl_matrix_get(muy, 0, 0) + gsl_ran_gaussian(seed, sYd));
                } else {
                    gsl_matrix_set(Yd, 0, n, (fre_1(xnd, fd, mud, wd) / s2u + gsl_matrix_get(muy, 0, 0) / s2Y) /
                                             (1 / s2Y + 1 / s2u) +
                                             gsl_ran_gaussian(seed, sqrt(1 / (1 / s2Y + 1 / s2u))));
                }
            }
            gsl_matrix_free(muy);
            break;

        case 'p':
            //positive real-valued observations Eq. (15)
            muy = gsl_matrix_calloc(1, 1);
            for (int n = 0; n < N; n++) {
                xnd = gsl_matrix_get(X, d, n);
                Zn = gsl_matrix_submatrix(Z, 0, n, K, 1);
                Bd_view = gsl_matrix_submatrix(Bd, 0, 0, K, 1);
                matrix_multiply(&Zn.matrix, &Bd_view.matrix, muy, 1, 0, CblasTrans, CblasNoTrans);
                if (xnd == missing || gsl_isnan(xnd)) {
                    gsl_matrix_set(Yd, 0, n, gsl_matrix_get(muy, 0, 0) + gsl_ran_gaussian(seed, sYd));
                } else {
                    gsl_matrix_set(Yd, 0, n, (f_1(xnd, fd, mud, wd) / s2u + gsl_matrix_get(muy, 0, 0) / s2Y) /
                                             (1 / s2Y + 1 / s2u) +
                                             gsl_ran_gaussian(seed, sqrt(1 / (1 / s2Y + 1 / s2u))));
                }
            }
            gsl_matrix_free(muy);
            break;

        case 'n':
            //count observations
            muy = gsl_matrix_calloc(1, 1);
            for (int n = 0; n < N; n++) {
                xnd = gsl_matrix_get(X, d, n);
                Zn = gsl_matrix_submatrix(Z, 0, n, K, 1);
                Bd_view = gsl_matrix_submatrix(Bd, 0, 0, K, 1);
                matrix_multiply(&Zn.matrix, &Bd_view.matrix, muy, 1, 0, CblasTrans, CblasNoTrans);
                if (xnd == missing || gsl_isnan(xnd)) {
                    gsl_matrix_set(Yd, 0, n, gsl_matrix_get(muy, 0, 0) + gsl_ran_gaussian(seed, sYd));
                } else {
                    gsl_matrix_set(Yd, 0, n, truncnormrnd(gsl_matrix_get(muy, 0, 0), sYd, f_1(xnd, fd, mud, wd),
                                                          f_1(xnd + 1, fd, mud, wd), seed));
                }
                if (isinf(gsl_matrix_get(Yd, 0, n)) || isnan(gsl_matrix_get(Yd, 0, n))) {
                    LOG(OUTPUT_NORMAL,
                        "EXECUTION STOPPED: the distribution of attribute %d (%d in Matlab) leads to numerical errors at the sampler. \n                   Have you considered applying a pre-processing transformation to this attribute? \n",
                        d, d + 1);
                    break;
                }
            }
            gsl_matrix_free(muy);
            break;

        case 'c':
            //categorical observations
            muy = gsl_matrix_calloc(1, Rd);
            for (int n = 0; n < N; n++) {
                xnd = gsl_matrix_get(X, d, n);
                Zn = gsl_matrix_submatrix(Z, 0, n, K, 1);
                Bd_view = gsl_matrix_submatrix(Bd, 0, 0, K, Rd);
                //muy=Zn * Bd_view
                matrix_multiply(&Zn.matrix, &Bd_view.matrix, muy, 1, 0, CblasTrans, CblasNoTrans);
                if (xnd == missing || gsl_isnan(xnd)) {
                    for (int r = 0; r < Rd; r++) {
                        gsl_matrix_set(Yd, r, n, gsl_matrix_get(muy, 0, r) + gsl_ran_gaussian(seed, sYd));
                    }
                } else {
                    double maxY = 0;
                    double ytrue = gsl_matrix_get(Yd, xnd - 1, n);
                    for (int r = 0; r < Rd; r++) {
                        double ydr = gsl_matrix_get(Yd, r, n);
                        if ((ydr != ytrue) & (ydr > maxY)) { maxY = ydr; }
                    }
                    gsl_matrix_set(Yd, xnd - 1, n,
                                   truncnormrnd(gsl_matrix_get(muy, 0, xnd - 1), sYd, maxY, GSL_POSINF, seed));
                    for (int r = 0; r < Rd; r++) {
                        if (r != xnd - 1) {
                            gsl_matrix_set(Yd, r, n, truncnormrnd(gsl_matrix_get(muy, 0, r), sYd, GSL_NEGINF,
                                                                  gsl_matrix_get(Yd, xnd - 1, n), seed));
                        }
                    }
                }
            }
            gsl_matrix_free(muy);
            break;

        case 'b':
            //binary observations
            muy = gsl_matrix_calloc(1, 1);
            for (int n = 0; n < N; n++) {
                xnd = (int) gsl_matrix_get(X, d, n);
                Zn = gsl_matrix_submatrix(Z, 0, n, K, 1);
                Bd_view = gsl_matrix_submatrix(Bd, 0, 0, K, 1);
                matrix_multiply(&Zn.matrix, &Bd_view.matrix, muy, 1, 0, CblasTrans, CblasNoTrans);
                if (xnd == -1 || gsl_isnan(xnd)) {//missing data
                    gsl_matrix_set(Yd, 0, n, gsl_matrix_get(muy, 0, 0) + gsl_ran_gaussian(seed, sYd));
                } else if (xnd == 0) {
                    gsl_matrix_set(Yd, 0, n, truncnormrnd(gsl_matrix_get(muy, 0, 0), sYd, GSL_NEGINF, 0, seed));
                } else if (xnd == 1) {
                    gsl_matrix_set(Yd, 0, n, truncnormrnd(gsl_matrix_get(muy, 0, 0), sYd, 0, GSL_POSINF, seed));
                } else {
                    LOG(OUTPUT_NORMAL, "Error! xnd for binary is not 0, 1, -1 (for missing data)");
                }
            }
            gsl_matrix_free(muy);
            break;

        case 'o':
            //ordinal observations
            gsl_vector *Ymax = gsl_vector_calloc(Rd);
            gsl_vector *Ymin = gsl_vector_alloc(Rd);
            gsl_vector_set_all(Ymin, GSL_POSINF);
            muy = gsl_matrix_calloc(1, 1);
            for (int n = 0; n < N; n++) {
                xnd = gsl_matrix_get(X, d, n);
                Zn = gsl_matrix_submatrix(Z, 0, n, K, 1);
                Bd_view = gsl_matrix_submatrix(Bd, 0, 0, K, 1);
                //muy=Zn * Bd_view
                matrix_multiply(&Zn.matrix, &Bd_view.matrix, muy, 1, 0, CblasTrans, CblasNoTrans);
                if (xnd == missing || gsl_isnan(xnd)) {
                    gsl_matrix_set(Yd, 0, n, gsl_matrix_get(muy, 0, 0) + gsl_ran_gaussian(seed, sYd));
                } else if (xnd == 1) {
                    gsl_matrix_set(Yd, 0, n, truncnormrnd(gsl_matrix_get(muy, 0, 0), sYd, GSL_NEGINF,
                                                          gsl_vector_get(thetad, xnd - 1), seed));
                    if (gsl_matrix_get(Yd, 0, n) > gsl_vector_get(Ymax, xnd - 1)) {
                        gsl_vector_set(Ymax, xnd - 1, gsl_matrix_get(Yd, 0, n));
                    }
                    if (gsl_matrix_get(Yd, 0, n) < gsl_vector_get(Ymin, xnd - 1)) {
                        gsl_vector_set(Ymin, xnd - 1, gsl_matrix_get(Yd, 0, n));
                    }
                } else {
                    gsl_matrix_set(Yd, 0, n,
                                   truncnormrnd(gsl_matrix_get(muy, 0, 0), sYd, gsl_vector_get(thetad, xnd - 2),
                                                gsl_vector_get(thetad, xnd - 1), seed));
                    if (gsl_matrix_get(Yd, 0, n) > gsl_vector_get(Ymax, xnd - 1)) {
                        gsl_vector_set(Ymax, xnd - 1, gsl_matrix_get(Yd, 0, n));
                    }
                    if (gsl_matrix_get(Yd, 0, n) < gsl_vector_get(Ymin, xnd - 1)) {
                        gsl_vector_set(Ymin, xnd - 1, gsl_matrix_get(Yd, 0, n));
                    }
                }
            }
            gsl_matrix_free(muy);
            //Sample Theta
            for (int r = 1; r < Rd - 1; r++) {
                double xlo;
                double xhi;
                if (gsl_vector_get(thetad, r) > gsl_vector_get(Ymax, r)) { xlo = gsl_vector_get(thetad, r); }
                else { xlo = gsl_vector_get(Ymax, r); }
                if (gsl_vector_get(thetad, r + 1) < gsl_vector_get(Ymin, r + 1)) {
                    xhi = gsl_vector_get(thetad, r + 1);
                } else { xhi = gsl_vector_get(Ymin, r + 1); }
                //theta_r^d=Gaussian(theta_r^d|0,sigma_{theta}^2)I(theta_r^d>theta_{r-1}^d)
                gsl_vector_set(thetad, r, truncnormrnd(0, stheta, xlo, xhi, seed));
            }
            break;
    }
}

// Sample Rho : pseudo-observation of the adjacency matrix
void SampleRho(double missing,
               int N,
               int K,
               char Ca,
               double fa,
               double s2Rho,
               double s2u,
               gsl_matrix *A,
               gsl_matrix *Z,
               gsl_matrix *vecRho,
               gsl_matrix *H,
               const gsl_rng *seed) {
    double sRho = sqrt(s2Rho);
    gsl_matrix_view Zview = gsl_matrix_submatrix(Z, 0, 0, K, N);
    gsl_matrix_view H_view = gsl_matrix_submatrix(H, 0, 0, K, K);
    gsl_matrix *vecH = gsl_matrix_calloc(K * K, 1);
    gsl_matrix2vector(vecH, &H_view.matrix);
    gsl_matrix *mu_rho;
    gsl_matrix *aux = gsl_matrix_calloc(1, K * K);
    gsl_matrix *ZmT = gsl_matrix_calloc(1, K);
    gsl_matrix *ZnT = gsl_matrix_calloc(1, K);
    gsl_matrix_view Zn;
    gsl_matrix_view Zm;
    int a_nm;
    // Sample pseudo adjacency matrix
    if (Ca == 'w') {
        double mud;
        double wd;
        //https://gist.github.com/microo8/4065693
        gsl_vector_view An_view;

        for (int m = 0; m < N; m++) {
            Zm = gsl_matrix_submatrix(Z, 0, m, K, 1);
            An_view = gsl_matrix_row(A, m);
            mud = compute_vector_mean(N, missing, &An_view.vector);
            wd = 1. / sqrt(compute_vector_var(N, missing, &An_view.vector));
            gsl_matrix_transpose_memcpy(ZmT, &Zm.matrix);
            for (int n = 0; n < m; n++) {//try to keep Rho matrix symmetric
                mu_rho = gsl_matrix_calloc(1, 1);
                a_nm = gsl_matrix_get(A, m, n);
                Zn = gsl_matrix_submatrix(Z, 0, n, K, 1);
                gsl_matrix_transpose_memcpy(ZnT, &Zn.matrix);
                gsl_matrix_transpose_memcpy(ZmT, &Zm.matrix);
                //gsl_Kronecker_product(aux, &z_n_column.matrix, &z_m_column.matrix);//???
                gsl_Kronecker_product(aux, ZnT, ZmT);
                matrix_multiply(aux, vecH, mu_rho, 1, 0, CblasNoTrans, CblasNoTrans);
                if (gsl_isnan(a_nm) || a_nm == missing) {
                    gsl_matrix_set(vecRho, m * N + n, 0, gsl_matrix_get(mu_rho, 0, 0) + gsl_ran_gaussian(seed, sRho));
                } else {
                    gsl_matrix_set(vecRho, m * N + n, 0,
                                   (f_w(a_nm, fa, mud, wd) / s2u + gsl_matrix_get(mu_rho, 0, 0) / s2Rho) /
                                   (1 / s2Rho + 1 / s2u) + gsl_ran_gaussian(seed, sqrt(1 / (1 / s2Rho + 1 / s2u))));
                }
                gsl_matrix_set(vecRho, n * N + m, 0,
                               gsl_matrix_get(vecRho, m * N + n, 0)); //extend symmetric matrix Rho to its vector

            }
        }

    } else if (Ca == 'b') {
        // binary values
        for (int m = 0; m < N; m++) {
            Zm = gsl_matrix_submatrix(Z, 0, m, K, 1);
            for (int n = 0; n < m; n++) {
                mu_rho = gsl_matrix_calloc(1, 1);
                Zn = gsl_matrix_submatrix(Z, 0, n, K, 1);
                gsl_matrix_transpose_memcpy(ZnT, &Zn.matrix);
                gsl_matrix_transpose_memcpy(ZmT, &Zm.matrix);
                gsl_Kronecker_product(aux, ZnT, ZmT);
                matrix_multiply(aux, vecH, mu_rho, 1, 0, CblasNoTrans, CblasNoTrans);

                a_nm = (int) gsl_matrix_get(A, m, n);
                if (gsl_isnan(a_nm || a_nm == missing)) {
                    gsl_matrix_set(vecRho, m * N + n, 0, gsl_matrix_get(mu_rho, 0, 0) + gsl_ran_gaussian(seed, sRho));
                } else if (a_nm == 0) {

                    gsl_matrix_set(vecRho, m * N + n, 0,
                                   truncnormrnd(gsl_matrix_get(mu_rho, 0, 0), sRho, GSL_NEGINF, 0, seed));
                } else if (a_nm == 1) {

                    gsl_matrix_set(vecRho, m * N + n, 0,
                                   truncnormrnd(gsl_matrix_get(mu_rho, 0, 0), sRho, 0, GSL_POSINF, seed));
                }
                gsl_matrix_set(vecRho, n * N + m, 0, gsl_matrix_get(vecRho, m * N + n, 0));

                //print the problematic part of the code
                if (isinf(gsl_matrix_get(vecRho, m * N + n, 0))) {
                    LOG(OUTPUT_DEBUG, "mu: %3.2f\n", gsl_matrix_get(mu_rho, 0, 0));
                    LOG(OUTPUT_DEBUG, "\n vec(H):\n");
                    for (int row = 0; row < K * K; ++row)
                        LOG(OUTPUT_DEBUG, "%6.5f\t", gsl_matrix_get(vecH, row, 0));
                    LOG(OUTPUT_DEBUG, "\n Z x Z:\n");
                    for (int col = 0; col < K * K; ++col)
                        LOG(OUTPUT_DEBUG, "%6.5f\t", gsl_matrix_get(aux, 0, col));
                    LOG(OUTPUT_DEBUG, "\n---\n---\n");
                    LOG(OUTPUT_DEBUG, "m:%d , n:%d , A_{mn}: %.2f, A_{mn}: %d, Rho: %.3f\n", m, n,
                        gsl_matrix_get(A, m, n), a_nm,
                        gsl_matrix_get(vecRho, m * N + n, 0));
                }
            }
        }
    }
    gsl_matrix_free(ZmT);
    gsl_matrix_free(ZnT);
    gsl_matrix_free(vecH);
    gsl_matrix_free(aux);
    gsl_matrix_free(mu_rho);
}

double SampleAlpha(int Kplus, int N, const gsl_rng *seed) {
    double Harmonic_N = 0.;
    double i = 1.;
    while (i < N + 1) {
        Harmonic_N += 1.0 / i;
        i++;
    }

    // equation 21 https://arxiv.org/pdf/1011.6293.pdf
    double alpha = gsl_ran_gamma(seed, 1 + Kplus,
                                 1 / (1 + Harmonic_N));
    return alpha;
}

double Samples2Y(double missing, int N, int d, int K, char Cd, int Rd, double fd, double mud, double wd, double s2u,
                 double s2theta, gsl_matrix *X, gsl_matrix *Z, gsl_matrix *Yd, gsl_matrix *Bd, gsl_vector *thetad,
                 const gsl_rng *seed) {
    double a = 2;
    double b = 2;
    gsl_matrix_view Zn;
    gsl_matrix_view Bd_view;
    gsl_matrix *muy;
    double sumY = 0;
    double xnd;

    muy = gsl_matrix_calloc(1, 1);

    for (int n = 0; n < N; n++) {
        for (int r = 0; r < Rd; r++) {
            xnd = gsl_matrix_get(X, d, n);
            Zn = gsl_matrix_submatrix(Z, 0, n, K, 1);
            Bd_view = gsl_matrix_submatrix(Bd, 0, 0, K, 1);
            //muy=Zn*Bd_view
            matrix_multiply(&Zn.matrix, &Bd_view.matrix, muy, 1, 0, CblasTrans, CblasNoTrans);
            sumY += pow(gsl_matrix_get(Yd, 0, n) - gsl_matrix_get(muy, 0, 0), 2);
        }
    }
    gsl_matrix_free(muy);
    //Eq. (18) of Infinite Sparse Factor Analysis and Infinite Independent Components Analysis
    //IG(s2Y|a+ND/2,b/(1+b/2*tr(E^T E)))
    double precision = gsl_ran_gamma(seed, a + N / 2., 1 / (b + sumY / 2));//inverse Gamma prior
    return 1. / precision;
}

// sample noise variance of the pseudo-observation of the adjacency matrix
double
Samples2Rho(int N, int K, gsl_matrix *A, gsl_matrix *Z, gsl_matrix *vecRho, gsl_matrix *vecH, const gsl_rng *seed) {
    double a = 1;
    double b = 1;

    gsl_matrix *aux = gsl_matrix_calloc(N * N, 1);
    gsl_matrix *S = gsl_matrix_calloc(K * K, N * N);
    gsl_matrix *D = gsl_matrix_calloc(1, 1);

    gsl_matrix_view Z_view = gsl_matrix_submatrix(Z, 0, 0, K, N);


    gsl_Kronecker_product(S, &Z_view.matrix, &Z_view.matrix);
    matrix_multiply(S, vecH, aux, 1, 0, CblasTrans, CblasNoTrans);
    gsl_matrix_sub(aux, vecRho);
    matrix_multiply(aux, aux, D, 1, 0, CblasTrans, CblasNoTrans);

    for (int n = 0; n < N * N; n++) {
        if (isinf(gsl_matrix_get(vecRho, n, 0))) {
            LOG(OUTPUT_DEBUG, "%d , %.3f \n", n, gsl_matrix_get(vecRho, n, 0));
        }
    }
    LOG(OUTPUT_DEBUG, "sample s2rho: %.4f\n", gsl_matrix_get(D, 0, 0));
    double precision = gsl_ran_gamma(seed, a + N * N / 2., 1 / (b + gsl_matrix_get(D, 0, 0) / 2.));//???????
    gsl_matrix_free(aux);
    gsl_matrix_free(S);
    gsl_matrix_free(D);
    return 1. / precision;
}


double Samples2H(int K, gsl_matrix *vecH, const gsl_rng *seed) {
    double a = 2;
    double b = 1;
    gsl_matrix *var = gsl_matrix_calloc(1, 1);

    matrix_multiply(vecH, vecH, var, 1, 0, CblasTrans, CblasNoTrans);
    LOG(OUTPUT_DEBUG, "sample s2H: %.4f\n", gsl_matrix_get(var, 0, 0));
    double precision = gsl_ran_gamma(seed, a + K * K / 2., b / (1 + b * gsl_matrix_get(var, 0, 0) / 2));
    gsl_matrix_free(var);
    return 1. / precision;
}


int IBPsampler_func(double missing,
                    gsl_matrix *X,
                    char *C,
                    char *Net,//*** the type of network
                    gsl_matrix *Z, //The binary feature vector
                    gsl_matrix **B, //the weighting vectors D * maxK * 1, initially all 0
                    gsl_vector **theta,
                    gsl_matrix *H,// The homophily matrix
                    gsl_matrix *A,// The adjacency matrix
                    int *R, //unordered index set of the categorical data
                    double *f, //mapping function from the real space R into the observation space
                    double fa,
                    double *mu, // mean, mu[d] = mean(X[d]) the mean value of an attribute to all nodes
                    double *w, //variance
                    int maxR,
                    int bias,
                    int N,
                    int D,
                    int K,
                    double alpha,
                    double s2B,
                    double *s2Y,
                    double s2Rho,
                    double s2H,
                    double s2u,
                    int maxK,
                    int Nsim) {

    LOG(OUTPUT_NORMAL, "N=%d, D=%d, K=%d", N, D, K);
    LOG(OUTPUT_INFO, "Running inference algorithm (currently inside C++ routine...)");

    double s2theta = 2;

    gsl_rng *seed = gsl_rng_alloc(gsl_rng_taus);
    time_t clck = time(nullptr);
    gsl_rng_set(seed, clck);

    // auxiliary variables
    int Kest = K;
    gsl_matrix *P = gsl_matrix_calloc(maxK, maxK);
    gsl_matrix_set_identity(P);
    gsl_matrix_view P_view = gsl_matrix_submatrix(P, 0, 0, Kest, Kest);

    gsl_matrix_view Z_view = gsl_matrix_submatrix(Z, 0, 0, Kest, N);
    matrix_multiply(&Z_view.matrix, &Z_view.matrix, &P_view.matrix, 1, 1 / s2B, CblasNoTrans, CblasTrans);
    gsl_matrix *Pnon = gsl_matrix_calloc(maxK, maxK);


    // Initialize Q and Qnon
    gsl_matrix *Q = gsl_matrix_calloc(maxK * maxK, maxK * maxK);
    double ldet_Q = 0;


    double coeff = s2Rho / s2H;
    gsl_matrix_view Q_view_init = gsl_matrix_submatrix(Q, 0, 0, Kest * Kest, Kest * Kest);
    compute_inverse_Q_directly(N, Kest, &Z_view.matrix, coeff, &Q_view_init.matrix);

    gsl_matrix *Qnon = gsl_matrix_calloc(maxK * maxK, maxK * maxK);
    double ldet_Q_n = 0;



    // initialize counts
    int *nest = new int[maxK];
    for (int i = 0; i < maxK; i++) {
        nest[i] = 0;
    }

    for (int k = 0; k < Kest; k++) {
        int ncount = 0;
        for (int n = 0; n < N; n++) {
            if (gsl_matrix_get(Z, k, n) == 1) { ncount++; }
        }
        nest[k] = ncount;
    }

    auto **Y = (gsl_matrix **) calloc(D, sizeof(gsl_matrix *));
    auto **lambda = (gsl_matrix **) calloc(D, sizeof(gsl_matrix *));
    auto **lambdanon = (gsl_matrix **) calloc(D, sizeof(gsl_matrix *));

    //Initialize Y
    for (int d = 0; d < D; d++) {
        switch (C[d]) {
            double xnd;
            case 'g':
                Y[d] = gsl_matrix_calloc(1, N);
                for (int n = 0; n < N; n++) {
                    xnd = gsl_matrix_get(X, d, n);
                    if (xnd == missing || gsl_isnan(xnd)) {
                        //  if the real observation is missing, use random number follow Gaussian distribution with mean 0 and std sY
                        //  mean 0 because Bd is initially all 0
                        gsl_matrix_set(Y[d], 0, n, gsl_ran_gaussian(seed, sqrt(s2Y[d])));
                    } else {
                        //  f[d] is meaningless in this function call, it set pseudo ob to w * (x - mu)
                        gsl_matrix_set(Y[d], 0, n, fre_1(xnd, f[d], mu[d], w[d]));
                    }
                }

                break;

            case 'p':
                Y[d] = gsl_matrix_calloc(1, N);
                for (int n = 0; n < N; n++) {
                    xnd = gsl_matrix_get(X, d, n);

                    if (xnd == missing || gsl_isnan(xnd)) {
                        gsl_matrix_set(Y[d], 0, n, gsl_ran_gaussian(seed, sqrt(s2Y[d])));
                    } else {
                        gsl_matrix_set(Y[d], 0, n, f_1(xnd, f[d], mu[d], w[d])); //+gsl_ran_gaussian (seed, s2Y)
                    }
                }
                break;

            case 'n':
                Y[d] = gsl_matrix_calloc(1, N);
                lambda[d] = gsl_matrix_calloc(maxK, 1);
                matrix_multiply(Z, Y[d], lambda[d], 1, 0, CblasNoTrans, CblasTrans);
                for (int n = 0; n < N; n++) {
                    xnd = gsl_matrix_get(X, d, n);

                    if (xnd == missing || gsl_isnan(xnd)) {
                        gsl_matrix_set(Y[d], 0, n, gsl_ran_gaussian(seed, sqrt(s2Y[d])));
                    } else {
                        gsl_matrix_set(Y[d], 0, n, f_1(xnd, f[d], mu[d],
                                                       w[d]));// +gsl_ran_beta (seed, 5,1)????? shouldn't this be floor (f_1(xnd,f[d], mu[d], w[d]))
                    }
                }
                break;

            case 'c':
                Y[d] = gsl_matrix_calloc(R[d], N);
                for (int n = 0; n < N; n++) {
                    xnd = gsl_matrix_get(X, d, n);
                    if (xnd == missing || gsl_isnan(xnd)) {
                        for (int r = 0; r < R[d]; r++) {
                            gsl_matrix_set(Y[d], r, n, gsl_ran_gaussian(seed, sqrt(s2Y[d])));
                        }
                    } else {
                        gsl_matrix_set(Y[d], xnd - 1, n, truncnormrnd(0, sqrt(s2Y[d]), 0, GSL_POSINF, seed));
                        for (int r = 0; r < R[d]; r++) {
                            if (r != xnd - 1) {
                                gsl_matrix_set(Y[d], r, n, truncnormrnd(0, sqrt(s2Y[d]), GSL_NEGINF,
                                                                        gsl_matrix_get(Y[d], xnd - 1, n), seed));
                            }
                        }
                    }
                }
                break;

            case 'b':
                Y[d] = gsl_matrix_calloc(1, N);
                for (int n = 0; n < N; n++) {
                    xnd = (int) gsl_matrix_get(X, d, n);
                    if (xnd == -1 || gsl_isnan(xnd)) {
                        // it is a missing binary value
                        gsl_matrix_set(Y[d], 0, n, gsl_ran_gaussian(seed, sqrt(s2Y[d])));

                    } else if (xnd == 0) {
                        // it just gives it a negative number follows normal distribution with mean 0
                        gsl_matrix_set(Y[d], 0, n, truncnormrnd(0, sqrt(s2Y[d]), GSL_NEGINF, 0, seed));
                    } else if (xnd == 1) {
                        gsl_matrix_set(Y[d], 0, n, truncnormrnd(0, sqrt(s2Y[d]), 0, GSL_POSINF, seed));
                    }
                }
                break;

            case 'o':
                Y[d] = gsl_matrix_calloc(R[d], N);
                gsl_vector_view Xd_view = gsl_matrix_row(X, d);
                double maxX = compute_vector_max(N, missing, &Xd_view.vector);
                gsl_vector_set(theta[d], 0, -sqrt(s2Y[d]));
                for (int r = 1; r < R[d] - 1; r++) {
                    gsl_vector_set(theta[d], r,
                                   gsl_vector_get(theta[d], r - 1) + (4 * sqrt(s2Y[d]) / maxX) * rand01());
                }
                gsl_vector_set(theta[d], R[d] - 1, GSL_POSINF);
                for (int n = 0; n < N; n++) {
                    xnd = gsl_matrix_get(X, d, n);

                    if (xnd == missing || gsl_isnan(xnd)) {
                        gsl_matrix_set(Y[d], 0, n, gsl_ran_gaussian(seed, sqrt(s2Y[d])));
                    } else if (xnd == 1) {
                        gsl_matrix_set(Y[d], 0, n,
                                       truncnormrnd(0, sqrt(s2Y[d]), GSL_NEGINF, gsl_vector_get(theta[d], xnd - 1), seed));
                    } else {
                        gsl_matrix_set(Y[d], 0, n, truncnormrnd(0, sqrt(s2Y[d]), gsl_vector_get(theta[d], xnd - 2),
                                                                gsl_vector_get(theta[d], xnd - 1), seed));
                    }
                }
                break;
        }
        // R[d] is always 1
        lambda[d] = gsl_matrix_calloc(maxK, R[d]);
        matrix_multiply(Z, Y[d], lambda[d], 1, 0, CblasNoTrans, CblasTrans);
        lambdanon[d] = gsl_matrix_calloc(maxK, R[d]);
    }


    LOG(OUTPUT_INFO, "Initialize Rho using pseudo-observation of the adjacency matrix ..... !\n");
    gsl_matrix *Rho = gsl_matrix_calloc(N, N);
    gsl_matrix *vecRho = gsl_matrix_calloc(N * N, 1);


    if (Net[0] == 'w') {
        double a_mn;
        double mu_a;
        double wa;
        gsl_vector_view Am_view;
        for (int m = 0; m < N; m++) {
            Am_view = gsl_matrix_row(A, m);
            //mu_a  = compute_vector_mean(N, missing, &Am_view.vector);
            mu_a = compute_vector_min(N, missing, &Am_view.vector) - 1e-6;
            wa = 1. / sqrt(compute_vector_var(N, missing, &Am_view.vector));
            for (int n = 0; n < m; n++) {
                a_mn = gsl_matrix_get(A, m, n);
                if ((a_mn == missing) || gsl_isnan(a_mn)) {
                    gsl_matrix_set(vecRho, m * N + n, 0, gsl_ran_gaussian(seed, sqrt(s2Rho)));
                } else {
                    gsl_matrix_set(vecRho, m * N + n, 0, f_w(a_mn, fa, mu_a, wa));
                }
                gsl_matrix_set(vecRho, n * N + m, 0, gsl_matrix_get(vecRho, m * N + n, 0));
            }
        }
    } else if (Net[0] == 'b') {
        // adjacency matrix is binary
        int a_mn;
        for (int m = 0; m < N; m++) {
            for (int n = 0; n < m; n++) {
                a_mn = (int) gsl_matrix_get(A, m, n);
                if ((a_mn == missing) || gsl_isnan(a_mn)) {
                    // use 0 as the mean here because H is currently all 0, thus, mean is 0
                    gsl_matrix_set(vecRho, m * N + n, 0, gsl_ran_gaussian(seed, sqrt(s2Rho)));
                } else if (a_mn == 0) {
                    // it just give it a negative number follows normal distribution with mean 0
                    gsl_matrix_set(vecRho, m * N + n, 0, truncnormrnd(0, sqrt(s2Rho), GSL_NEGINF, 0, seed));
                } else if (a_mn == 1) {
                    gsl_matrix_set(vecRho, m * N + n, 0, truncnormrnd(0, sqrt(s2Rho), 0, GSL_POSINF, seed));
                }
                gsl_matrix_set(vecRho, n * N + m, 0, gsl_matrix_get(vecRho, m * N + n, 0));
            }
        }
    }

    // compute full Eta
    gsl_matrix *Eta = gsl_matrix_calloc(maxK * maxK, 1);
    gsl_matrix_view Eta_init_view = gsl_matrix_submatrix(Eta, 0, 0, Kest * Kest, 1);

    gsl_matrix *ZoZ = gsl_matrix_calloc(Kest * Kest, N * N);

    gsl_Kronecker_product(ZoZ, &Z_view.matrix, &Z_view.matrix);
    matrix_multiply(ZoZ, vecRho, &Eta_init_view.matrix, 1, 0, CblasNoTrans, CblasNoTrans);
    gsl_matrix_free(ZoZ);

    // todo debug only
    print_matrix(Eta, "init Eta", maxK);


    gsl_matrix *Etanon = gsl_matrix_calloc(maxK * maxK, 1);


    LOG(OUTPUT_DEBUG, "Before IT loop...");
    LOG(OUTPUT_DEBUG, "Nsim = %d", Nsim);

    // main loop
    for (int it = 0; it < Nsim; it++) {
        LOG(OUTPUT_NORMAL, "Start iteration %d", it);
        gsl_vector2matrix(vecRho, Rho);
        int Kaux = AcceleratedGibbs(maxK, bias, N, D, Kest, C, R, alpha, s2B, s2Y, s2H, s2Rho, Y, Rho, vecRho, Z, nest,
                                    P, Pnon, lambda, lambdanon, Q, Qnon, Eta, Etanon, &ldet_Q, &ldet_Q_n);

        LOG(OUTPUT_NORMAL, "iteration %d, K= %d\n", it, Kaux);


        if (Kaux == 0) { return Kest; } else { Kest = Kaux; }

        P_view = gsl_matrix_submatrix(P, 0, 0, Kest, Kest);
        gsl_matrix *S = gsl_matrix_calloc(Kest, Kest);
        gsl_matrix_memcpy(S, &P_view.matrix);
        inverse(S, Kest);
        gsl_matrix *MuB = gsl_matrix_calloc(Kest, 1);

        for (int d = 0; d < D; d++) {
            //Sample Bs
            if (C[d] == 'c') {
                gsl_vector_view Bd_view;
                for (int r = 0; r < R[d] - 1; r++) {
                    gsl_matrix_view L_view = gsl_matrix_submatrix(lambda[d], 0, r, Kest, 1);
                    matrix_multiply(S, &L_view.matrix, MuB, 1, 0, CblasNoTrans, CblasNoTrans);
                    Bd_view = gsl_matrix_subcolumn(B[d], r, 0, Kest);
                    gsl_vector_view MuB_view = gsl_matrix_column(MuB, 0);
                    mvnrnd(&Bd_view.vector, S, &MuB_view.vector, Kest, seed);
                }
                Bd_view = gsl_matrix_subcolumn(B[d], R[d] - 1, 0, Kest);
                gsl_vector_set_zero(&Bd_view.vector);
            } else {
                gsl_matrix_view Lnon_view = gsl_matrix_submatrix(lambda[d], 0, 0, Kest, 1);
                matrix_multiply(S, &Lnon_view.matrix, MuB, 1, 0, CblasNoTrans, CblasNoTrans);

                gsl_vector_view Bd_view = gsl_matrix_subcolumn(B[d], 0, 0, Kest);
                gsl_vector_view MuB_view = gsl_matrix_subcolumn(MuB, 0, 0, Kest);
                mvnrnd(&Bd_view.vector, S, &MuB_view.vector, Kest, seed);
            }

            //Sample Y
            SampleY(missing, N, d, Kest, C[d], R[d], f[d], mu[d], w[d], s2Y[d], s2u, s2theta, X, Z, Y[d], B[d],
                    theta[d], seed);
            if (C[d] != 'c' && C[d] != 'o') {
                double aux = Samples2Y(missing, N, d, Kest, C[d], R[d], f[d], mu[d], w[d], s2u, s2theta, X, Z, Y[d],
                                       B[d], theta[d], seed);
                if (aux != 0 && !isinf(aux) && !isnan(aux)) {
                    s2Y[d] = aux;
                } else {
                    return Kest;
                }
            }

            //Update lambda
            matrix_multiply(Z, Y[d], lambda[d], 1, 0, CblasNoTrans, CblasTrans);

        }

        // Sample Hs
        gsl_matrix_view H_view = gsl_matrix_submatrix(H, 0, 0, Kest, Kest);

        LOG(OUTPUT_DEBUG, "old H");
        for (int i = 0; i < Kest; i++) {
            for (int j = 0; j < Kest; j++) {
                LOG(OUTPUT_DEBUG, "%3.2f", gsl_matrix_get(&H_view.matrix, i, j));
            }
            LOG(OUTPUT_DEBUG, "")
        }
        gsl_matrix *MuH = gsl_matrix_calloc(Kest * Kest, 1);

        gsl_matrix *vecH = gsl_matrix_calloc(Kest * Kest, 1);
        gsl_matrix2vector(vecH, &H_view.matrix);
        gsl_vector_view vecH_view = gsl_matrix_subcolumn(vecH, 0, 0, Kest * Kest);
        LOG(OUTPUT_DEBUG, "vecH_view size = %zd", (&vecH_view.vector)->size);


        gsl_matrix_view Q_view = gsl_matrix_submatrix(Q, 0, 0, Kest * Kest, Kest * Kest);
        gsl_matrix_view Eta_view = gsl_matrix_submatrix(Eta, 0, 0, Kest * Kest, 1);

        //  MuH sometime is very large causing new H become too large
        //  MuH = Q * S^T * vec(rho) = Q * Eta  (see equation 14)
        matrix_multiply(&Q_view.matrix, &Eta_view.matrix, MuH, 1, 0, CblasNoTrans, CblasNoTrans);
        gsl_vector_view MuH_view = gsl_matrix_column(MuH, 0);

        mvnrnd(&vecH_view.vector, &Q_view.matrix, &MuH_view.vector, Kest * Kest, seed);

        gsl_vector2matrix(vecH, &H_view.matrix);

        print_matrix(&Eta_view.matrix, "Eta matrix", Kest);
        print_matrix(&H_view.matrix, "new H");
        print_matrix(MuH, "Mu H");
        print_matrix(&Q_view.matrix, "Q matrix");


        // *****End Sampling Hs
        // sampleRho

        SampleRho(missing, N, Kest, Net[0], fa, s2Rho, s2u, A, Z, vecRho, &H_view.matrix, seed);
        // sample the variance of Rho and H
        s2Rho = Samples2Rho(N, Kest, A, Z, vecRho, vecH, seed);
        s2H = Samples2H(Kest, vecH, seed);

        alpha = SampleAlpha(Kest, N, seed);

        LOG(OUTPUT_INFO, "\n");
        LOG(OUTPUT_INFO, "s2_rho --> %.3f", s2Rho);
        LOG(OUTPUT_INFO, "s2_h   --> %.3f", s2H);
        LOG(OUTPUT_INFO, "alpha  --> %.3f", alpha);


        LOG(OUTPUT_INFO, "\n\nB matrix");
        for (int i = 0; i < D; i++) {
            gsl_matrix *Brow = B[i];
            for (int j = 0; j < Kest; j++) {
                if (OUTPUT_LEVEL >= OUTPUT_INFO) {
                    cout << gsl_matrix_get(Brow, j, 0) << " , ";
                }
            }
            LOG(OUTPUT_INFO, "");
        }
        LOG(OUTPUT_INFO, "\n");


        gsl_matrix_free(vecH);
        gsl_matrix_free(MuH);
        gsl_matrix_free(S);
    }
    LOG(OUTPUT_DEBUG, "After IT loop...\n");

    for (int d = 0; d < D; d++) {
        gsl_matrix_free(Y[d]);
        gsl_matrix_free(lambda[d]);
        gsl_matrix_free(lambdanon[d]);
    }
    free(lambda);
    free(lambdanon);
    free(Y);

    gsl_matrix_free(P);
    gsl_matrix_free(Pnon);
    gsl_matrix_free(Q);
    gsl_matrix_free(Qnon);
    gsl_matrix_free(Rho);
    gsl_matrix_free(vecRho);
    gsl_matrix_free(Eta);
    gsl_matrix_free(Etanon);

    delete[] nest;
    return Kest;
}


int initialize_func(int N,
                    int D,
                    int maxK,
                    double missing,
                    gsl_matrix *X,
                    const char *C,
                    gsl_matrix **B,
                    gsl_vector **theta,
                    int *R,
                    double *f,
                    double *mu,
                    double *w,
                    double *s2Y) {

    int maxR = 1;
    auto *maxX = new double[D];
    auto *minX = new double[D];
    auto *meanX = new double[D];
    auto *varX = new double[D];
    gsl_vector_view Xd_view;
    for (int d = 0; d < D; d++) {
        Xd_view = gsl_matrix_row(X, d);
        maxX[d] = compute_vector_max(N, missing, &Xd_view.vector);
        minX[d] = compute_vector_min(N, missing, &Xd_view.vector);
        meanX[d] = compute_vector_mean(N, missing, &Xd_view.vector);
        varX[d] = compute_vector_var(N, missing, &Xd_view.vector);
        mu[d] = 1;
        R[d] = 1;
        w[d] = 1;
        switch (C[d]) {
            case 'g':
                s2Y[d] = 2;
                B[d] = gsl_matrix_calloc(maxK, 1);
                mu[d] = meanX[d];
                if (varX[d] > 0) { w[d] = 1 / sqrt(varX[d]); }
                else { w[d] = 1; }
                break;
            case 'p':
                s2Y[d] = 2;
                B[d] = gsl_matrix_calloc(maxK, 1);
                mu[d] = minX[d] - 1e-6;
                if (varX[d] > 0) { w[d] = 1 / sqrt(varX[d]); }
                else { w[d] = 1; }
                break;
            case 'n':
                s2Y[d] = 2;
                B[d] = gsl_matrix_calloc(maxK, 1);
                mu[d] = minX[d] - 1;
                if (varX[d] > 0) { w[d] = 1 / sqrt(varX[d]); }
                else { w[d] = 1; }
                break;
            case 'c':
                s2Y[d] = 1;
                R[d] = (int) maxX[d];
                B[d] = gsl_matrix_calloc(maxK, R[d]);
                if (R[d] > maxR) { maxR = R[d]; }
                break;
            case 'o':
                s2Y[d] = 1;
                R[d] = (int) maxX[d];
                B[d] = gsl_matrix_calloc(maxK, 1);
                theta[d] = gsl_vector_alloc(R[d]);
                if (R[d] > maxR) { maxR = R[d]; }
                break;
            case 'b':
                s2Y[d] = 1;
                R[d] = 1;
                B[d] = gsl_matrix_calloc(maxK, 1);
                break;
        }
    }

    delete[] maxX;
    delete[] minX;
    delete[] meanX;
    delete[] varX;
    return maxR;
}
